#include "hip/hip_runtime.h"
#include <av_planning/gpu_calculator.h>
#include <av_planning/planning_comm.h>
#include <vector>
#include <iostream>
/*#include <boost/serialization/vector.hpp>*/
/*#include <boost/archive/text_oarchive.hpp>*/
/*#include <boost/archive/text_iarchive.hpp>*/

using namespace std;
using namespace av_planning;
using namespace av_msgs;

namespace{
    const DTYPE eps = 1e-6;
    const int C_LEN = 6;
    const int CDOT_LEN = 5;
    const int CDDOT_LEN = 4;
    const int CURVE_OFFSET = C_LEN+CDOT_LEN+CDDOT_LEN+1;
    const int SPLINE_FIXLEN = 7;
    const int SPLINE_OFFSET = 5;
    const int POSE_OFFSET = 7; //x, y, timestamp
    const int RANGE_OFFSET = 2;
}

void getTrajectoryCost(const std::vector<av_msgs::Pose>& poses, const std::vector<double>& d_list, const Range& d_range, const std::vector<double>& accelerations, const std::vector<double>& jerks, const ManeuverState& maneuver_state, Cost& cost, double acceleration_normalization_, double jerk_normalization_, double curvature_normalization_) {
    double velocity_cost_sum=0, acceleration_cost_sum=0, jerk_cost_sum=0, curvature_cost_sum=0, position_cost_sum=0;
    int size = poses.size();
    for(int i=0; i<size; i++) {
        Pose pose = poses[i];
        velocity_cost_sum += abs(pose.velocity-maneuver_state.v);
        acceleration_cost_sum += abs(accelerations[i]);
        jerk_cost_sum += abs(jerks[i]);
        curvature_cost_sum += abs(pose.curvature);
        position_cost_sum += abs(d_list[i]);
    }
    double velocity_cost = velocity_cost_sum/size;
    double acceleration_cost = acceleration_cost_sum/size;
    double jerk_cost = jerk_cost_sum/size;
    double curvature_cost = curvature_cost_sum/size;
    double position_cost = position_cost_sum/size;
    double similarity_cost = 0; // the similarity cost will be computed in trajectory selection

    // Normalize the costs
    if(maneuver_state.v > eps) {
        velocity_cost = velocity_cost / abs(maneuver_state.v);
    } else{
        ; // occurs in stop mode
    }
    cost.velocity_cost = velocity_cost;
    cost.acceleration_cost = acceleration_cost / acceleration_normalization_;
    cost.jerk_cost = jerk_cost / jerk_normalization_;
    cost.curvature_cost = curvature_cost / curvature_normalization_;
    cost.position_cost = position_cost / fabs(d_range.max_value - d_range.min_value);
    cost.similarity_cost = similarity_cost;
}

static void HandleError( hipError_t err, const char *file, int line){
    if (err != hipSuccess){
        printf( "%s in %s at line %d\n", hipGetErrorString( err ), file, line);
        exit( EXIT_FAILURE);
    }
}
#define HANDLE_ERROR( err ) (HandleError(err, __FILE__, __LINE__))

void toArrayCurves(const std::vector<Curve>& s_curves, DTYPE* s_curves_list){
    int tmp_index = 0;
    for (vector<Curve>::const_iterator iter = s_curves.begin(); iter != s_curves.end(); iter++){
        /*assert(iter->c.size() == C_LEN);*/
        /*assert(iter->c_dot.size() == CDOT_LEN);*/
        /*assert(iter->c_ddot.size() == CDDOT_LEN);*/
        for (int i = 0; i < C_LEN - iter->c.size(); i++){
            s_curves_list[tmp_index++] = 0;
        }
        for (vector<double>::const_iterator son_iter = iter->c.begin(); son_iter != iter->c.end(); son_iter++){
            s_curves_list[tmp_index++] = *son_iter;
        }
        for (int i = 0; i < CDOT_LEN - iter->c_dot.size(); i++){
            s_curves_list[tmp_index++] = 0;
        }
        for (vector<double>::const_iterator son_iter = iter->c_dot.begin(); son_iter != iter->c_dot.end(); son_iter++){
            s_curves_list[tmp_index++] = *son_iter;
        }
        for (int i = 0; i < CDDOT_LEN - iter->c_ddot.size(); i++){
            s_curves_list[tmp_index++] = 0;
        }
        for (vector<double>::const_iterator son_iter = iter->c_ddot.begin(); son_iter != iter->c_ddot.end(); son_iter++){
            s_curves_list[tmp_index++] = *son_iter;
        }
        s_curves_list[tmp_index++] = iter->terminal;
    }
}

void toArrayTlist(const std::vector<double>& t_list, DTYPE* t_array){
    int tmp_index = 0;
    for (vector<double>::const_iterator iter = t_list.begin(); iter != t_list.end(); iter++){
        t_array[tmp_index++] = *iter;
    }
}

void toArraySpline(const spline& ref_curve, DTYPE* spline_array, int spline_size){
    int tmp_index = 0;
    spline_array[tmp_index++] = ref_curve.m_b0;
    spline_array[tmp_index++] = ref_curve.m_c0;
    spline_array[tmp_index++] = ref_curve.m_left;
    spline_array[tmp_index++] = ref_curve.m_right;
    spline_array[tmp_index++] = ref_curve.m_left_value;
    spline_array[tmp_index++] = ref_curve.m_right_value;
    spline_array[tmp_index++] = ref_curve.m_force_linear_extrapolation;
    for(int i = 0; i < spline_size; i++){
        spline_array[tmp_index++] = ref_curve.m_x[i];
        spline_array[tmp_index++] = ref_curve.m_y[i];
        spline_array[tmp_index++] = ref_curve.m_a[i];
        spline_array[tmp_index++] = ref_curve.m_b[i];
        spline_array[tmp_index++] = ref_curve.m_c[i];
    }
}

__device__ DTYPE polyVal(DTYPE x, DTYPE* coefficients, int coef_index, int coef_len){
	DTYPE y = 0;
	for(int i=0; i<coef_len; i++) {
		y += (coefficients[coef_index+i]*powf(x,(coef_len-i-1)));
	}
	return y;
}


__device__ int lower_bound(DTYPE *array, int size, DTYPE key, int index){
    int first = 0, middle;
    int half, len;
    len = size;
    while(len > 0) {
        half = len >> 1;
        middle = first + half;
        if(array[middle*SPLINE_OFFSET+SPLINE_FIXLEN+index] < key) {     
            first = middle + 1;          
            len = len-half-1;       
        }
        else
            len = half;           
    }
    return first;
}

__device__ DTYPE splineCurveVal(DTYPE x, DTYPE* spline, int spline_size){
    int match_id = lower_bound(spline, spline_size, x, 0) - 1;
    if (match_id < 0) match_id = 0;
    if (match_id >= spline_size) match_id = spline_size-1;
    DTYPE h = x - spline[match_id*SPLINE_OFFSET+SPLINE_FIXLEN];
    DTYPE interpol;
    if(x < spline[0*SPLINE_OFFSET+SPLINE_FIXLEN])
        interpol = (spline[0]*h + spline[1])*h + spline[0*SPLINE_OFFSET+SPLINE_FIXLEN+1];
    else if(x > spline[(spline_size-1)*SPLINE_OFFSET+SPLINE_FIXLEN])
        interpol = (spline[(spline_size-1)*SPLINE_OFFSET+SPLINE_FIXLEN+3]*h + spline[(spline_size-1)*SPLINE_OFFSET+SPLINE_FIXLEN+4])*h 
            + spline[(spline_size-1)*SPLINE_OFFSET+SPLINE_FIXLEN+1];
    else
        interpol = ((spline[match_id*SPLINE_OFFSET+SPLINE_FIXLEN+2]*h+spline[match_id*SPLINE_OFFSET+SPLINE_FIXLEN+3])*h
            + spline[match_id*SPLINE_OFFSET+SPLINE_FIXLEN+4])*h + spline[match_id*SPLINE_OFFSET+SPLINE_FIXLEN+1];
    return interpol;
}

__device__ DTYPE splineCurveDeriv1(DTYPE x, DTYPE* spline, int spline_size){
    int match_id = lower_bound(spline, spline_size, x, 0) - 1;
    if (match_id < 0) match_id = 0;
    if (match_id >= spline_size) match_id = spline_size-1;
    DTYPE h = x - spline[match_id*SPLINE_OFFSET+SPLINE_FIXLEN];
    DTYPE interpol;
    if(x < spline[0*SPLINE_OFFSET+SPLINE_FIXLEN])
        interpol = 2*spline[0]*h + spline[1];
    else if(x > spline[(spline_size-1)*SPLINE_OFFSET+SPLINE_FIXLEN])
        interpol = 2*spline[(spline_size-1)*SPLINE_OFFSET+SPLINE_FIXLEN+3]*h + spline[(spline_size-1)*SPLINE_OFFSET+SPLINE_FIXLEN+4];
    else
        interpol = (3*spline[match_id*SPLINE_OFFSET+SPLINE_FIXLEN+2]*h+2*spline[match_id*SPLINE_OFFSET+SPLINE_FIXLEN+3])*h +
            spline[match_id*SPLINE_OFFSET+SPLINE_FIXLEN+4];
    return interpol;
}

__device__ DTYPE splineCurveDeriv2(DTYPE x, DTYPE* spline, int spline_size){
    int match_id = lower_bound(spline, spline_size, x, 0) - 1;
    if (match_id < 0) match_id = 0;
    if (match_id >= spline_size) match_id = spline_size-1;
    DTYPE h = x - spline[match_id*SPLINE_OFFSET+SPLINE_FIXLEN];
    DTYPE interpol;
    if(x < spline[0*SPLINE_OFFSET+SPLINE_FIXLEN])
        interpol = 2*spline[0];
    else if(x > spline[(spline_size-1)*SPLINE_OFFSET+SPLINE_FIXLEN])
        interpol = 2*spline[(spline_size-1)*SPLINE_OFFSET+SPLINE_FIXLEN+3];
    else
        interpol = 6*spline[match_id*SPLINE_OFFSET+SPLINE_FIXLEN+2]*h+2*spline[match_id*SPLINE_OFFSET+SPLINE_FIXLEN+3];
    return interpol;
}

__device__ DTYPE polyCurveVal(DTYPE cur_t, DTYPE* curves, int index, bool is_extend){
    DTYPE endx = curves[index*CURVE_OFFSET+CURVE_OFFSET-1];
    DTYPE res = 0;
    if( cur_t <= endx ){
        res = polyVal(cur_t, curves, index*CURVE_OFFSET, C_LEN);
    }else if(is_extend){
        DTYPE v = polyVal(endx, curves, index*CURVE_OFFSET+C_LEN, CDOT_LEN);
        DTYPE t = cur_t - endx;
        res = polyVal(endx, curves, index*CURVE_OFFSET, C_LEN) + v * t;
    }else{
        res = polyVal(endx, curves, index*CURVE_OFFSET, C_LEN);
    }
    return res;
}

__device__ DTYPE polyCurveDeriv1(DTYPE cur_t, DTYPE* curves, int index){
    DTYPE endx = curves[index*CURVE_OFFSET+CURVE_OFFSET-1];
    DTYPE res = 0;
    if( cur_t <= endx ){
        res = polyVal(cur_t, curves, index*CURVE_OFFSET+C_LEN, CDOT_LEN);
    }else{
        res = polyVal(endx, curves, index*CURVE_OFFSET+C_LEN, CDOT_LEN);
    }
    return res;
}

__device__ DTYPE polyCurveDeriv2(DTYPE cur_t, DTYPE* curves, int index){
    DTYPE endx = curves[index*CURVE_OFFSET+CURVE_OFFSET-1];
    DTYPE res = 0;
    if( cur_t <= endx ){
        res = polyVal(cur_t, curves, index*CURVE_OFFSET+C_LEN+CDOT_LEN, CDDOT_LEN);
    }else{
        res = polyVal(endx, curves, index*CURVE_OFFSET+C_LEN+CDOT_LEN, CDDOT_LEN);
    }
    return res;
}

__global__ void kernel(DTYPE* s_curves, DTYPE* d_curves, DTYPE* t_list, DTYPE* ref_x_curve, DTYPE* ref_y_curve, DTYPE* ref_theta_curve, DTYPE* s0, DTYPE* range, int* t_list_len, int* spline_xy_size, int* spline_theta_size, DTYPE* sample_t, 
        DTYPE* s_dot_list, DTYPE* s_ddot_list, DTYPE* d_list, DTYPE* pose_list, bool* is_valid){
    __shared__ DTYPE d0;
    if(threadIdx.x == 0){
        DTYPE s_real = polyCurveVal(t_list[0], s_curves, blockIdx.x, true);
        d0 = polyCurveVal(s_real-*s0, d_curves, blockIdx.y, false);
    }
    is_valid[blockIdx.x*gridDim.y+blockIdx.y] = 1;
    __syncthreads();
    //s
    DTYPE cur_t = t_list[threadIdx.x];
    DTYPE s_real = polyCurveVal(cur_t, s_curves, blockIdx.x, true);
    //s_real
    DTYPE s = s_real - *s0;
    //s_dot
    DTYPE s_dot = polyCurveDeriv1(cur_t, s_curves, blockIdx.x);
    //s_ddot
    DTYPE s_ddot = polyCurveDeriv2(cur_t, s_curves, blockIdx.x);
    //d
    DTYPE d = polyCurveVal(s, d_curves, blockIdx.y, false);
    //d_dot
    DTYPE d_dot = polyCurveDeriv1(s, d_curves, blockIdx.y);
    //d_ddot
    DTYPE d_ddot = polyCurveDeriv2(s, d_curves, blockIdx.y);

    DTYPE d_min = (d0 < range[0] ? d0 : range[0]);
    DTYPE d_max = (d0 > range[1] ? d0 : range[1]);
    if(d < d_min || d > d_max){
        is_valid[blockIdx.x*gridDim.y+blockIdx.y] = 0;
    }

    DTYPE ref_x = splineCurveVal(s_real, ref_x_curve, *spline_xy_size);
    DTYPE ref_y = splineCurveVal(s_real, ref_y_curve, *spline_xy_size);
    DTYPE ref_theta = splineCurveVal(s_real, ref_theta_curve, *spline_theta_size);
    DTYPE ref_x_dot = splineCurveDeriv1(s_real, ref_x_curve, *spline_xy_size);
    DTYPE ref_y_dot = splineCurveDeriv1(s_real, ref_y_curve, *spline_xy_size);
    DTYPE ref_theta_dot = splineCurveDeriv1(s_real, ref_theta_curve, *spline_theta_size);
    DTYPE ref_x_ddot = splineCurveDeriv2(s_real, ref_x_curve, *spline_xy_size);
    DTYPE ref_y_ddot = splineCurveDeriv2(s_real, ref_y_curve, *spline_xy_size);
    DTYPE ref_theta_ddot = splineCurveDeriv2(s_real, ref_theta_curve, *spline_theta_size);
    DTYPE sin_ref_theta = sinf(ref_theta);
    DTYPE cos_ref_theta = cosf(ref_theta);
    DTYPE x = ref_x + d*sin_ref_theta;
    DTYPE y = ref_y - d*cos_ref_theta;
    DTYPE x_dot = ref_x_dot + d_dot * sin_ref_theta + d * cos_ref_theta * ref_theta_dot;
    DTYPE y_dot = ref_y_dot - d_dot * cos_ref_theta + d * sin_ref_theta * ref_theta_dot;
    DTYPE x_ddot = ref_x_ddot + d_ddot * sin_ref_theta + d_dot * cos_ref_theta * ref_theta_dot + d_dot * cos_ref_theta * ref_theta_dot - d * sin_ref_theta * ref_theta_dot * ref_theta_dot + d * cos_ref_theta * ref_theta_ddot;
    DTYPE y_ddot = ref_y_ddot - d_ddot * cos_ref_theta + d_dot * sin_ref_theta * ref_theta_dot + d_dot * sin_ref_theta * ref_theta_dot + d * cos_ref_theta * ref_theta_dot * ref_theta_dot + d * sin_ref_theta * ref_theta_ddot;
    DTYPE theta = atan2f(y_dot, x_dot);
    DTYPE vel = s_dot;
    DTYPE acc = s_ddot;
    DTYPE curvature = (x_dot * y_ddot - y_dot * x_ddot) / powf(x_dot * x_dot + y_dot * y_dot, 1.5);
    int pos_save_index = ((blockIdx.x*gridDim.y+blockIdx.y)*(*t_list_len)+threadIdx.x)*POSE_OFFSET;
    int dot_save_index = ((blockIdx.x*gridDim.y+blockIdx.y)*(*t_list_len)+threadIdx.x);
    pose_list[pos_save_index+0] = *sample_t*threadIdx.x;
    pose_list[pos_save_index+1] = x;
    pose_list[pos_save_index+2] = y;
    pose_list[pos_save_index+3] = theta;
    pose_list[pos_save_index+4] = vel;
    pose_list[pos_save_index+5] = acc;
    pose_list[pos_save_index+6] = curvature;
    s_dot_list[dot_save_index] = s_dot;
    s_ddot_list[dot_save_index] = s_ddot;
    d_list[dot_save_index] = d;
}

void GpuCalculator::combineTrajectoriesGpu(const std::vector<double>& t_list_, double sample_t_, const ManeuverState &maneuver_state, const std::vector<Curve>& s_curves, const std::vector<Curve>& d_curves, const spline& ref_x_curve,const spline& ref_y_curve, const spline& ref_theta_curve, const av_msgs::Localize& localize, const double& s0, Range d_range, 
    std::vector<av_msgs::Trajectory >& trajectories, std::vector<Cost>& costs, double acceleration_normalization_, double jerk_normalization_, double curvature_normalization_) {

    trajectories.clear();
    costs.clear();

    uint64_t start = Now<uint64_t, std::nano>();
    const int s_curves_size = s_curves.size();
    const int d_curves_size = d_curves.size();
    const int t_size = t_list_.size();
    assert(ref_x_curve.m_x.size() == ref_x_curve.m_y.size());
    const int ref_xy_size = ref_x_curve.m_x.size();
    const int ref_theta_size = ref_theta_curve.m_x.size();
    const int ref_mem_xy_size = ref_xy_size*SPLINE_OFFSET + SPLINE_FIXLEN;
    const int ref_mem_theta_size = ref_theta_size*SPLINE_OFFSET + SPLINE_FIXLEN;
    const DTYPE sample_t = sample_t_;
    const DTYPE s0_f = s0;
    /*const int traj_num = s_curves_size*d_curves_size;*/
	/*trajectories.resize(traj_num);*/
	/*costs.resize(traj_num);*/
    /*cout << d_curves.size() << " " << d_curves[0].c_ddot.size() << " " << d_curves[40].c[1] << " " << d_curves[90].c_dot.size() <<endl;*/

    DTYPE* s_curves_list = new DTYPE[s_curves_size*(CURVE_OFFSET)];
    DTYPE* d_curves_list = new DTYPE[d_curves_size*(CURVE_OFFSET)];
    DTYPE* t_array = new DTYPE[t_size];
    DTYPE* ref_x_array = new DTYPE[ref_mem_xy_size];
    DTYPE* ref_y_array = new DTYPE[ref_mem_xy_size];
    DTYPE* ref_theta_array = new DTYPE[ref_mem_theta_size];
    DTYPE* d_range_list = new DTYPE[RANGE_OFFSET];

    toArrayCurves(s_curves, s_curves_list);
    toArrayCurves(d_curves, d_curves_list);
    toArrayTlist(t_list_, t_array);
    /*ref_x_curve.toArray(ref_x_array, ref_xy_size); */
    /*ref_y_curve.toArray(ref_y_array, ref_xy_size); */
    /*ref_theta_curve.toArray(ref_theta_array, ref_theta_size); */
    toArraySpline(ref_x_curve, ref_x_array, ref_xy_size);
    toArraySpline(ref_y_curve, ref_y_array, ref_xy_size);
    toArraySpline(ref_theta_curve, ref_theta_array, ref_theta_size);
    d_range_list[0] = d_range.min_value;
    d_range_list[1] = d_range.max_value;

    DTYPE* dev_s_curves;
    DTYPE* dev_d_curves;
    DTYPE* dev_t_array;
    DTYPE* dev_ref_x_curve;
    DTYPE* dev_ref_y_curve;
    DTYPE* dev_ref_theta_curve;
    DTYPE* dev_sample_t;
    DTYPE* dev_s0;
    DTYPE* dev_range;
    int* dev_t_array_len;
    int* dev_spline_xy_size;
    int* dev_spline_theta_size;
    //output
    DTYPE* dev_s_dot_list;
    DTYPE* dev_s_ddot_list;
    DTYPE* dev_d_list;
    DTYPE* dev_pose_list;
    bool* dev_is_valid;

    //input
    hipMalloc( (void**)&dev_s_curves, s_curves_size*CURVE_OFFSET*sizeof(DTYPE) );
    hipMalloc( (void**)&dev_d_curves, d_curves_size*CURVE_OFFSET*sizeof(DTYPE) );
    hipMalloc( (void**)&dev_t_array, t_size*sizeof(DTYPE) );
    hipMalloc( (void**)&dev_ref_x_curve, ref_mem_xy_size*sizeof(DTYPE) );
    hipMalloc( (void**)&dev_ref_y_curve, ref_mem_xy_size*sizeof(DTYPE) );
    hipMalloc( (void**)&dev_ref_theta_curve, ref_mem_theta_size*sizeof(DTYPE) );
    hipMalloc( (void**)&dev_s0, sizeof(DTYPE));
    hipMalloc( (void**)&dev_range, RANGE_OFFSET*sizeof(DTYPE));
    hipMalloc( (void**)&dev_t_array_len, sizeof(int) );
    hipMalloc( (void**)&dev_spline_xy_size, sizeof(int) );
    hipMalloc( (void**)&dev_spline_theta_size, sizeof(int) );
    hipMalloc( (void**)&dev_sample_t, sizeof(DTYPE) );
    //output
    hipMalloc( (void**)&dev_s_dot_list, t_size*s_curves_size*d_curves_size*sizeof(DTYPE) );
    hipMalloc( (void**)&dev_s_ddot_list, t_size*s_curves_size*d_curves_size*sizeof(DTYPE) );
    hipMalloc( (void**)&dev_d_list, t_size*s_curves_size*d_curves_size*sizeof(DTYPE) );
    hipMalloc( (void**)&dev_pose_list, t_size*s_curves_size*d_curves_size*POSE_OFFSET*sizeof(DTYPE) );
    hipMalloc( (void**)&dev_is_valid, s_curves_size*d_curves_size*sizeof(bool) );

    hipMemcpy( dev_s_curves, s_curves_list, s_curves_size*CURVE_OFFSET*sizeof(DTYPE), hipMemcpyHostToDevice );
    hipMemcpy( dev_d_curves, d_curves_list, d_curves_size*CURVE_OFFSET*sizeof(DTYPE), hipMemcpyHostToDevice );
    hipMemcpy( dev_t_array, t_array, t_size*sizeof(DTYPE), hipMemcpyHostToDevice );
    hipMemcpy( dev_ref_x_curve, ref_x_array, ref_mem_xy_size*sizeof(DTYPE), hipMemcpyHostToDevice );
    hipMemcpy( dev_ref_y_curve, ref_y_array, ref_mem_xy_size*sizeof(DTYPE), hipMemcpyHostToDevice );
    hipMemcpy( dev_ref_theta_curve, ref_theta_array, ref_mem_theta_size*sizeof(DTYPE), hipMemcpyHostToDevice );
    hipMemcpy( dev_s0, &s0_f, sizeof(DTYPE), hipMemcpyHostToDevice );
    hipMemcpy( dev_range, d_range_list, RANGE_OFFSET*sizeof(DTYPE), hipMemcpyHostToDevice );
    hipMemcpy( dev_t_array_len, &t_size, sizeof(int), hipMemcpyHostToDevice );
    hipMemcpy( dev_spline_xy_size, &ref_xy_size, sizeof(int), hipMemcpyHostToDevice );
    hipMemcpy( dev_spline_theta_size, &ref_theta_size, sizeof(int), hipMemcpyHostToDevice );
    hipMemcpy( dev_sample_t, &sample_t, sizeof(DTYPE), hipMemcpyHostToDevice );

    dim3 grids(s_curves_size, d_curves_size);
    kernel<<<grids, t_size>>>(dev_s_curves, dev_d_curves, dev_t_array, dev_ref_x_curve, dev_ref_y_curve, dev_ref_theta_curve, dev_s0, dev_range, dev_t_array_len, dev_spline_xy_size, dev_spline_theta_size, dev_sample_t, 
            dev_s_dot_list, dev_s_ddot_list, dev_d_list, dev_pose_list, dev_is_valid);

    DTYPE* pos_list = new DTYPE[t_size*s_curves_size*d_curves_size*POSE_OFFSET];
    DTYPE* s_dot_list = new DTYPE[t_size*s_curves_size*d_curves_size];
    DTYPE* s_ddot_list = new DTYPE[t_size*s_curves_size*d_curves_size];
    DTYPE* d_list = new DTYPE[t_size*s_curves_size*d_curves_size];
    bool* is_valid = new bool[s_curves_size*d_curves_size];
    HANDLE_ERROR(hipMemcpy( pos_list, dev_pose_list, t_size*s_curves_size*d_curves_size*POSE_OFFSET*sizeof(DTYPE), hipMemcpyDeviceToHost ));
    HANDLE_ERROR(hipMemcpy( s_dot_list, dev_s_dot_list, t_size*s_curves_size*d_curves_size*sizeof(DTYPE), hipMemcpyDeviceToHost ));
    HANDLE_ERROR(hipMemcpy( s_ddot_list, dev_s_ddot_list, t_size*s_curves_size*d_curves_size*sizeof(DTYPE), hipMemcpyDeviceToHost ));
    HANDLE_ERROR(hipMemcpy( d_list, dev_d_list, t_size*s_curves_size*d_curves_size*sizeof(DTYPE), hipMemcpyDeviceToHost ));
    HANDLE_ERROR(hipMemcpy( is_valid, dev_is_valid, s_curves_size*d_curves_size*sizeof(bool), hipMemcpyDeviceToHost ));
    /*for (int j = 0; j < POSE_OFFSET; j++)*/
        /*cout << pos_list[POSE_OFFSET*((0*d_curves_size+0)*t_size+5)+j] << " ";*/
    hipFree(dev_s_curves);
    hipFree(dev_d_curves);
    hipFree(dev_t_array);
    hipFree(dev_ref_x_curve);
    hipFree(dev_ref_y_curve);
    hipFree(dev_ref_theta_curve);
    hipFree(dev_s0);
    hipFree(dev_range);
    hipFree(dev_t_array_len);
    hipFree(dev_spline_xy_size);
    hipFree(dev_spline_theta_size);
    hipFree(dev_sample_t);
    hipFree(dev_s_dot_list);
    hipFree(dev_s_ddot_list);
    hipFree(dev_d_list);
    hipFree(dev_pose_list);
    hipFree(dev_is_valid);
    uint64_t end = Now<uint64_t, std::nano>();
    cout << end - start << endl;
    for(int i = 0; i < s_curves_size; i++){
        for(int j = 0; j < d_curves_size; j++){
            if(is_valid[i*d_curves_size+j]){
                Trajectory traj;
                vector<Pose> poses;
                vector<double> d_list_vec;
                vector<double> s_dot_list_vec;
                vector<double> s_ddot_list_vec;
                /*DTYPE d0 = d_list[(i*d_curves_size+j)*t_size+0];*/
                /*DTYPE d_min = (d0 < d_range.min_value ? d0 : d_range.min_value);*/
                /*DTYPE d_max = (d0 > d_range.max_value ? d0 : d_range.max_value);*/
                bool insert_flag = true;
                for(int k = 0; k < t_size; k++){
                    Pose pose;
                    int pose_index = ((i*d_curves_size+j)*t_size+k)*POSE_OFFSET;
                    pose.timestamp = pos_list[pose_index+0];
                    pose.x = pos_list[pose_index+1];
                    pose.y = pos_list[pose_index+2];
                    pose.theta = pos_list[pose_index+3];
                    pose.velocity = pos_list[pose_index+4];
                    pose.acceleration = pos_list[pose_index+5];
                    pose.curvature = pos_list[pose_index+6];
                    poses.push_back(pose);
                    int d_index = ((i*d_curves_size+j)*t_size+k);
                    /*if(d_list[d_index] < d_min || d_list[d_index] > d_max){*/
                        /*insert_flag = false;*/
                        /*break;*/
                    /*}*/
                    d_list_vec.push_back(d_list[d_index]);
                    s_dot_list_vec.push_back(s_dot_list[d_index]);
                    s_ddot_list_vec.push_back(s_ddot_list[d_index]);
                }
                if(insert_flag){
                    /*getTrajectoryInformation(poses, localize);*/
                    traj.poses = poses;
                    traj.driving_direction = 0;
                    traj.driving_state = 0;
			        Cost cost;
                    getTrajectoryCost(poses, d_list_vec, d_range, s_dot_list_vec, s_ddot_list_vec, maneuver_state, cost, acceleration_normalization_, jerk_normalization_, curvature_normalization_);
			        cost.time_cost = s_curves[i].terminal;
			        costs.push_back(cost);
			        trajectories.push_back(traj);
                }
            }
        }
    }
    delete[] s_curves_list;
    delete[] d_curves_list;
    delete[] t_array;
    delete[] ref_x_array;
    delete[] ref_y_array;
    delete[] ref_theta_array;
    delete[] d_range_list;

    delete[] pos_list;
    delete[] s_dot_list;
    delete[] s_ddot_list;
    delete[] d_list;
    delete[] is_valid;
}

